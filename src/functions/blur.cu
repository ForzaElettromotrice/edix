#include "hip/hip_runtime.h"
#include "blur.cuh"

__global__ void blur(const unsigned char *imgIn, unsigned char *imgOut, uint width, uint height, uint channels, int radius)
{
    int x = (int) (threadIdx.x + blockIdx.x * blockDim.x);
    int y = (int) (threadIdx.y + blockIdx.y * blockDim.y);

    if (x >= width || y >= height)
        return;

    int RGB[3];
    for (int k = 0; k < channels; ++k)
        RGB[k] = 0;

    int curr_i;
    int curr_j;
    int num = 0;

    for (int i = -radius; i <= radius; ++i)
    {
        for (int j = -radius; j <= radius; ++j)
        {
            curr_i = x + i;
            curr_j = y + j;
            if (curr_i < 0 || curr_j < 0 || curr_i >= width || curr_j >= height)
                continue;

            for (int k = 0; k < channels; ++k)
                RGB[k] += imgIn[(curr_i + curr_j * width) * channels + k];
            num++;
        }
    }

    for (int k = 0; k < channels; ++k)
    {
        RGB[k] /= num;
        imgOut[(x + y * width) * channels + k] = RGB[k];
    }


}
__global__ void blurShared(const unsigned char *imgIn, unsigned char *imgOut, uint width, uint height, uint channels, int radius)
{
    int absX = (int) (threadIdx.x + blockIdx.x * blockDim.x);
    int absY = (int) (threadIdx.y + blockIdx.y * blockDim.y);

    if (absX >= width || absY >= height)
        return;

    int relX = (int) threadIdx.x;
    int relY = (int) threadIdx.y;

    uint sSide = blockDim.x + 2 * radius;
    extern __shared__ unsigned char shared[];


//    int amount = (int) ((sSide + blockDim.x - 1) / blockDim.x);
//    int jump = (int) (blockDim.x * blockDim.y);
    int sX = (int) (absX - relX);
    int sY = (int) (absY - relY);

    if (relX == 0 && relY == 0)
    {
        for (int i = 0; i < sSide; i++)
        {
            for (int j = 0; j < sSide; j++)
            {
                if (sX + i - radius >= width || sY + j - radius >= height)
                    continue;

                for (int k = 0; k < channels; k++)
                    shared[(i + j * sSide) * channels + k] = imgIn[(sX - radius + i + (sY - radius + j) * width) * channels + k];
            }
        }
    }
    __syncthreads();

//    for (int n = 0; n < amount; ++n)
//    {
//        if (relX + relY * blockDim.x + n * jump >= pow(sSide, 2) || ((relX + n * jump) % sSide) + blockIdx.x * blockDim.x - radius >= width || ((relY + n * jump) / sSide) + blockIdx.y * blockDim.y - radius >= height)
//            continue;
//
//        for (int k = 0; k < channels; ++k)
//            shared[(relX + relY * blockDim.x + n * jump) * channels + k] = imgIn[(((relX + n * jump) % sSide) + blockIdx.x * blockDim.x - radius + (((relY + n * jump) / sSide) + blockIdx.y * blockDim.y - radius) * width) * channels + k];
//
//    }
//    __syncthreads();

    int RGB[3];
    for (int k = 0; k < channels; ++k)
        RGB[k] = 0;

    int x;
    int y;
    int num = 0;

    for (int i = -radius; i <= radius; ++i)
        for (int j = -radius; j <= radius; ++j)
        {
            x = relX + radius + i;
            y = relY + radius + j;

            if (absX + i < 0 || absX + i >= width || absY + j < 0 || absY + j >= height)
                continue;

            for (int k = 0; k < channels; ++k)
                RGB[k] += shared[(x + y * sSide) * channels + k];
            num++;
        }

    for (int k = 0; k < channels; ++k)
    {
        RGB[k] /= num;
        imgOut[(absX + absY * width) * channels + k] = RGB[k];
    }
}


unsigned char *blurSerial(const unsigned char *imgIn, uint width, uint height, uint channels, int radius, uint *oWidth, uint *oHeight)
{
    *oWidth = width;
    *oHeight = height;

    uint oSize = width * height * channels;
    auto imgOut = (unsigned char *) malloc(oSize * sizeof(unsigned char));
    if (imgOut == nullptr)
    {
        E_Print("Errore durante la malloc!\n");
        return nullptr;
    }

    int RGB[channels];
    memset(RGB, 0, channels * sizeof(int));

    int num;
    int curr_i;
    int curr_j;

    for (int x = 0; x < width; x++)
        for (int y = 0; y < height; y++)
        {
            num = 0;

            for (int i = -radius; i <= radius; i++)
                for (int j = -radius; j <= radius; j++)
                {
                    curr_i = x + i;
                    curr_j = y + j;
                    if ((curr_i < 0) || (curr_i >= width) || (curr_j < 0) || (curr_j >= height))
                        continue;

                    for (int k = 0; k < channels; ++k)
                        RGB[k] += imgIn[(curr_i + curr_j * width) * channels + k];
                    num++;
                }
            for (int k = 0; k < channels; ++k)
            {
                RGB[k] /= num;
                imgOut[(x + y * width) * channels + k] = RGB[k];
            }
        }


    return imgOut;
}
unsigned char *blurOmp(const unsigned char *imgIn, uint width, uint height, uint channels, int radius, uint *oWidth, uint *oHeight, int nThreads1, int nThreads2)
{
    *oWidth = width;
    *oHeight = height;

    uint oSize = width * height * channels;
    auto imgOut = (unsigned char *) malloc(oSize * sizeof(unsigned char));
    if (imgOut == nullptr)
    {
        E_Print("Errore durante la malloc!\n");
        return nullptr;
    }

    int RGB[channels];
    memset(RGB, 0, channels * sizeof(int));

    int num;
    int curr_i;
    int curr_j;

//TODO: magari cambiare schedule
#pragma omp parallel for num_threads(nThreads1) collapse(2) schedule(static) default(none) shared(width, height, channels, imgIn, imgOut, radius, nThreads2) private(RGB, num, curr_i, curr_j)
    for (int x = 0; x < width; x++)
        for (int y = 0; y < height; y++)
        {
            num = 0;
#pragma omp parallel for num_threads(nThreads2) collapse(2) schedule(static) default(none) shared(radius, width, height, channels, x, y, imgIn) private(curr_i, curr_j) reduction(+:num) reduction(+:RGB[:channels])
            for (int i = -radius; i <= radius; i++)
                for (int j = -radius; j <= radius; j++)
                {
                    curr_i = x + i;
                    curr_j = y + j;
                    if ((curr_i < 0) || (curr_i >= width) || (curr_j < 0) || (curr_j >= height))
                        continue;

                    for (int k = 0; k < channels; ++k)
                        RGB[k] += imgIn[(curr_i + curr_j * width) * channels + k];
                    num++;
                }
            for (int k = 0; k < channels; ++k)
            {
                RGB[k] /= num;
                imgOut[(x + y * width) * channels + k] = RGB[k];
            }
        }


    return imgOut;
}
unsigned char *blurCuda(const unsigned char *h_imgIn, uint width, uint height, uint channels, int radius, uint *oWidth, uint *oHeight, bool useShared)
{
    *oWidth = width;
    *oHeight = height;

    uint oSize = width * height * channels;
    auto h_imgOut = (unsigned char *) malloc(oSize * sizeof(unsigned char));
    if (h_imgOut == nullptr)
    {
        E_Print("Errore durante la malloc!\n");
        return nullptr;
    }
    mlock(h_imgIn, oSize * sizeof(unsigned char));


    unsigned char *d_imgIn;
    unsigned char *d_imgOut;
    hipMalloc(&d_imgIn, oSize * sizeof(unsigned char));
    hipMalloc(&d_imgOut, oSize * sizeof(unsigned char));

    hipMemcpy(d_imgIn, h_imgIn, oSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 gridSize = {(width + 7) / 8, (height + 7) / 8, 1};
    dim3 blockSize = {8, 8, 1};
    if (useShared)
    {
        auto sharedDim = (size_t) pow(8 + 2 * radius, 2) * channels;
        blurShared<<<gridSize, blockSize, sharedDim>>>(d_imgIn, d_imgOut, width, height, channels, radius);
    } else
        blur<<<gridSize, blockSize>>>(d_imgIn, d_imgOut, width, height, channels, radius);

    hipMemcpy(h_imgOut, d_imgOut, oSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    munlock(h_imgIn, oSize * sizeof(unsigned char));
    hipFree(d_imgIn);
    hipFree(d_imgOut);

    return h_imgOut;
}