#include "hip/hip_runtime.h"
//
// Created by f3m on 19/01/24.
//

#include "upscale.cuh"


void createSquare(unsigned char square[16][3], const unsigned char *img, int x, int y, uint width, uint height, uint channels)
{
    for (int i = -1; i < 3; ++i)
    {
        for (int j = -1; j < 3; ++j)
        {
            if (x - i < 0 || y - j < 0 || x + i >= width || y + j >= height)
                continue;
            for (int k = 0; k < channels; ++k)
                square[(i + 1) + (j + 1) * 4][k] = img[channels * (x + i + (y + j) * width) + k];
        }
    }
}

__global__ void bilinearUpscaleCUDA(const unsigned char *imgIn, unsigned char *imgOut, uint width, uint height, int factor)
{
    int x = (int) (threadIdx.x + blockIdx.x * blockDim.x);
    int y = (int) (threadIdx.y + blockIdx.y * blockDim.y);

    uint widthO = width * factor;
    uint heightO = height * factor;

    uint idx = x + y * widthO;

    int i;
    int j;
    int p00;
    int p01;
    int p10;
    int p11;
    double alpha;
    double beta;

    if (idx < widthO * heightO * 3)
    {
        i = x / factor;
        j = y / factor;
        alpha = ((double) x / factor) - i;
        beta = ((double) y / factor) - j;

        for (int k = 0; k < 3; k++)
        {
            p00 = imgIn[(i + j * width) * 3 + k];
            p01 = imgIn[(i + 1 + j * width) * 3 + k];
            p10 = imgIn[(i + (j + 1) * width) * 3 + k];
            p11 = imgIn[(i + 1 + (j + 1) * width) * 3 + k];

            imgOut[(idx * 3) + k] = (int) ((1 - alpha) * (1 - beta) * p00 + (1 - alpha) *
                                                                            beta * p01 + alpha * (1 - beta) * p10 +
                                           alpha * beta * p11);
        }
    }
}
__global__ void bicubicUpscaleCUDA(const unsigned char *imgIn, unsigned char *imgOut, uint width, uint height, int factor, uint channels)
{
    int absX = (int) (threadIdx.x + blockIdx.x * blockDim.x);
    int absY = (int) (threadIdx.y + blockIdx.y * blockDim.y);

    uint oWidth = width * factor;
    uint oHeight = height * factor;

    uint idx = absX + absY * oWidth;

    int i;
    int j;
    double alpha;
    double beta;
    unsigned char square[16][3];
    int pixel[3];

    if (idx < oWidth * oHeight * channels)
    {
        i = absX / factor;
        j = absY / factor;

        alpha = ((double) absX / factor) - i;
        beta = ((double) absY / factor) - j;


        createSquareDEVICE(square, imgIn, i, j, width, height, channels);

        for (int k = 0; k < channels; k++)
        {
            double p1 = cubicInterpolateDEVICE(square[0][k], square[1][k], square[2][k], square[3][k], alpha);
            double p2 = cubicInterpolateDEVICE(square[4][k], square[5][k], square[6][k], square[7][k], alpha);
            double p3 = cubicInterpolateDEVICE(square[8][k], square[9][k], square[10][k], square[11][k], alpha);
            double p4 = cubicInterpolateDEVICE(square[12][k], square[13][k], square[14 + k][k], square[15][k], alpha);
            double p = cubicInterpolateDEVICE(p1, p2, p3, p4, beta);

            if (p > 255)
                p = 255;
            else if (p < 0)
                p = 0;

            pixel[k] = (int) p;

        }

        imgOut[idx * channels] = pixel[0];
        imgOut[(idx * channels) + 1] = pixel[1];
        imgOut[(idx * channels) + 2] = pixel[2];
    }
}


int parseUpscaleArgs(char *args)
{
    char *pathIn = strtok(args, " ");
    char *pathOut = strtok(nullptr, " ");
    int factor = (int) strtol(strtok(nullptr, " "), nullptr, 10);

    if (pathIn == nullptr || pathOut == nullptr || factor == 0)
    {
        handle_error("usage " BOLD "funx upscale IN OUT FACTOR\n" RESET);
    }

    char *tpp = getStrFromKey((char *) "TPP");
    char *tup = getStrFromKey((char *) "TUP");
    uint width;
    uint height;
    uint channels;
    unsigned char *img = loadImage(pathIn, &width, &height, &channels);

    uint oWidth;
    uint oHeight;
    unsigned char *imgOut;

    if (strcmp(tpp, "Serial") == 0)
    {
        if (strcmp(tup, "Bilinear") == 0)
            imgOut = upscaleSerialBilinear(img, width, height, channels, factor, &oWidth, &oHeight);
        else if (strcmp(tup, "Bicubic") == 0)
            imgOut = upscaleSerialBicubic(img, width, height, channels, factor, &oWidth, &oHeight);
    } else if (strcmp(tpp, "OMP") == 0)
    {
        if (strcmp(tup, "Bilinear") == 0)
            imgOut = upscaleOmpBilinear(img, width, height, channels, factor, &oWidth, &oHeight, 4);
        else if (strcmp(tup, "Bicubic") == 0)
            imgOut = upscaleOmpBicubic(img, width, height, channels, factor, &oWidth, &oHeight, 4);
    } else if (strcmp(tpp, "CUDA") == 0)
    {
        if (strcmp(tup, "Bilinear") == 0)
            imgOut = upscaleCudaBilinear(img, width, height, channels, factor, &oWidth, &oHeight);
        else if (strcmp(tup, "Bicubic") == 0)
            imgOut = upscaleCudaBicubic(img, width, height, channels, factor, &oWidth, &oHeight);
    } else
    {
        free(img);
        free(tpp);
        handle_error("Invalid TPP\n");
    }

    if (imgOut != nullptr)
    {
        writeImage(pathOut, imgOut, oWidth, oHeight, channels);
        free(imgOut);
    }
    free(img);
    free(tpp);
    return 0;
}

unsigned char *upscaleSerialBilinear(const unsigned char *imgIn, uint width, uint height, uint channels, int factor, uint *oWidth, uint *oHeight)
{

    uint widthO = width * factor;
    uint heightO = height * factor;
    auto *imgOut = (unsigned char *) malloc((widthO * heightO * 3) * sizeof(unsigned char));
    if (imgOut == nullptr)
    {
        fprintf(stderr, RED "Error: " RESET "Error while malloc!\n");
        return nullptr;
    }

    int x;
    int y;
    int p00;
    int p01;
    int p10;
    int p11;
    double alpha;
    double beta;

    for (int i = 0; i < widthO; ++i)
    {
        for (int j = 0; j < heightO; ++j)
        {
            x = i / factor;
            y = j / factor;
            alpha = ((double) i / factor) - x;
            beta = ((double) j / factor) - y;

            for (int k = 0; k < channels; ++k)
            {
                p00 = imgIn[(x + y * width) * channels + k];
                p01 = x + 1 >= width ? p00 : imgIn[(x + 1 + y * width) * channels + k];
                p10 = y + 1 >= height ? p00 : imgIn[(x + (y + 1) * width) * channels + k];
                p11 = x + 1 >= width || y + 1 >= height ? p00 : imgIn[(x + 1 + (y + 1) * width) * channels + k];


                imgOut[(i + j * widthO) * channels + k] = bilinearInterpolation(p00, p01, p10, p11, alpha, beta);
            }
        }
    }

    *oWidth = widthO;
    *oHeight = heightO;

    return imgOut;
}
unsigned char *upscaleOmpBilinear(const unsigned char *imgIn, uint width, uint height, uint channels, int factor, uint *oWidth, uint *oHeight, int nThread)
{
    uint widthO = width * factor;
    uint heightO = height * factor;
    auto *imgOut = (unsigned char *) malloc((widthO * heightO * 3) * sizeof(unsigned char));
    if (imgOut == nullptr)
    {
        fprintf(stderr, RED "Error: " RESET "Error while malloc!\n");
        return nullptr;
    }

    int x;
    int y;
    int p00;
    int p01;
    int p10;
    int p11;
    double alpha;
    double beta;

#pragma omp parallel for num_threads(nThread) collapse(2) default(none) shared(imgIn, width, height, imgOut, widthO, heightO, factor, channels) private(x, y, alpha, beta, p00, p01, p10, p11)
    for (int i = 0; i < widthO; ++i)
    {
        for (int j = 0; j < heightO; ++j)
        {
            x = i / factor;
            y = j / factor;
            alpha = ((double) i / factor) - x;
            beta = ((double) j / factor) - y;

            for (int k = 0; k < channels; ++k)
            {
                //TODO: se sbordi, usa lo stesso pixel
                p00 = imgIn[(x + y * width) * 3 + k];
                p01 = imgIn[(x + 1 + y * width) * 3 + k];
                p10 = imgIn[(x + (y + 1) * width) * 3 + k];
                p11 = imgIn[(x + 1 + (y + 1) * width) * 3 + k];


                imgOut[(i + j * widthO) * 3 + k] = bilinearInterpolation(p00, p01, p10, p11, alpha, beta);
            }
        }
    }

    *oWidth = widthO;
    *oHeight = heightO;

    return imgOut;
}
unsigned char *upscaleCudaBilinear(const unsigned char *imgIn, uint width, uint height, uint channels, int factor, uint *oWidth, uint *oHeight)
{
    uint wf, hf;

    //host   
    wf = width * factor;
    hf = height * factor;

    uint iSize = width * height * 3;
    uint iSizeO = wf * hf * 3;

    auto h_imgOut = (unsigned char *) malloc(iSizeO * sizeof(unsigned char));
    if (h_imgOut == nullptr)
    {
        fprintf(stderr, RED "Error: " RESET "Errore nell'allocazione della memoria\n");
        munlock(imgIn, width * height * 3 * sizeof(unsigned char));
        return nullptr;
    }
    mlock(imgIn, iSize * sizeof(unsigned char));

    //device
    unsigned char *d_imgIn;
    unsigned char *d_imgOut;
    hipMalloc(&d_imgIn, iSize * sizeof(unsigned char));
    hipMalloc(&d_imgOut, iSizeO * sizeof(unsigned char));

    //copy
    hipMemcpy(d_imgIn, imgIn, iSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    //upscale
    dim3 gridSize = {(wf + 7) / 8, (hf + 7) / 8, 1};
    dim3 blockSize = {8, 8, 1};
    bilinearUpscaleCUDA<<<gridSize, blockSize>>>(d_imgIn, d_imgOut, width, height, factor);

    //copy back
    hipMemcpy(h_imgOut, d_imgOut, iSizeO * sizeof(unsigned char), hipMemcpyDeviceToHost);

    //free
    munlock(imgIn, iSize * sizeof(unsigned char));
    hipFree(d_imgIn);
    hipFree(d_imgOut);

    *oWidth = wf;
    *oHeight = hf;

    return h_imgOut;
}

unsigned char *upscaleSerialBicubic(const unsigned char *imgIn, uint width, uint height, uint channels, int factor, uint *oWidth, uint *oHeight)
{
    uint widthO = width * factor;
    uint heightO = height * factor;
    auto *imgOut = (unsigned char *) calloc(widthO * heightO * channels, sizeof(unsigned char));
    if (imgOut == nullptr)
    {
        fprintf(stderr, RED "Error: " RESET "Error while malloc!\n");
        return nullptr;
    }

    int x;
    int y;
    double alpha;
    double beta;
    unsigned char square[16][3];

    for (int i = 0; i < widthO; i++)
    {
        for (int j = 0; j < heightO; ++j)
        {
            x = i / factor;
            y = j / factor;

            alpha = ((double) i / factor) - x;
            beta = ((double) j / factor) - y;

            //TODO: i pixel mancanti devono essere la copia dell'originale
            createSquare(square, imgIn, x, y, width, height, channels);

            for (int k = 0; k < channels; k++)
            {
                double p1 = cubicInterpolate(square[0][k], square[1][k], square[2][k], square[3][k], alpha);
                double p2 = cubicInterpolate(square[4][k], square[5][k], square[6][k], square[7][k], alpha);
                double p3 = cubicInterpolate(square[8][k], square[9][k], square[10][k], square[11][k], alpha);
                double p4 = cubicInterpolate(square[12][k], square[13][k], square[14 + k][k], square[15][k], alpha);
                double p = cubicInterpolate(p1, p2, p3, p4, beta);

                if (p > 255)
                    p = 255;
                else if (p < 0)
                    p = 0;

                imgOut[(i + j * widthO) * channels + k] = (int) p;
            }
        }
    }


    *oWidth = widthO;
    *oHeight = heightO;

    return imgOut;
}
unsigned char *upscaleOmpBicubic(const unsigned char *imgIn, uint width, uint height, uint channels, int factor, uint *hWidth, uint *oHeight, int nThread)
{
    return nullptr;
}
unsigned char *upscaleCudaBicubic(const unsigned char *imgIn, uint width, uint height, uint channels, int factor, uint *oWidth, uint *oHeight)
{
    //init
    uint wf, hf;

    //host
    wf = width * factor;
    hf = height * factor;
    uint iSize = width * height * 3;
    mlock(imgIn, iSize * sizeof(unsigned char));
    uint iSizeO = wf * hf * 3;
    auto h_imgOut = (unsigned char *) malloc(iSizeO * sizeof(unsigned char));
    if (h_imgOut == nullptr)
    {
        fprintf(stderr, RED "Error: " RESET "Errore nell'allocazione della memoria\n");
        munlock(imgIn, iSize * sizeof(unsigned char));
        return nullptr;
    }

    //device
    unsigned char *d_imgIn;
    unsigned char *d_imgOut;
    hipMalloc(&d_imgIn, iSize * sizeof(unsigned char));
    hipMalloc(&d_imgOut, iSizeO * sizeof(unsigned char));

    //copy
    hipMemcpy(d_imgIn, imgIn, iSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    //upscale
    dim3 gridSize = {(wf + 7) / 8, (hf + 7) / 8, 1};
    dim3 blockSize = {8, 8, 1};
    bicubicUpscaleCUDA<<<gridSize, blockSize>>>(d_imgIn, d_imgOut, width, height, factor, channels);


    //copy back
    hipMemcpy(h_imgOut, d_imgOut, iSizeO * sizeof(unsigned char), hipMemcpyDeviceToHost);

    //free
    munlock(imgIn, iSize * sizeof(unsigned char));
    hipFree(d_imgIn);
    hipFree(d_imgOut);

    *oWidth = wf;
    *oHeight = hf;

    return h_imgOut;
}

int bilinearInterpolation(int p00, int p01, int p10, int p11, double alpha, double beta)
{
    return (int) ((1 - alpha) * (1 - beta) * p00 + (1 - alpha) * beta * p01 + alpha * (1 - beta) * p10 +
                  alpha * beta * p11);
}
double cubicInterpolate(double A, double B, double C, double D, double t)
{
    double a = -A / 2.0f + (3.0f * B) / 2.0f - (3.0f * C) / 2.0f + D / 2.0f;
    double b = A - (5.0f * B) / 2.0f + 2.0f * C - D / 2.0f;
    double c = -A / 2.0f + C / 2.0f;
    double d = B;
    return a * t * t * t + b * t * t + c * t + d;
//    return A + 0.5 * t * (C - A + t * (2.0 * A - 5.0 * B + 4.0 * C - D + t * (3.0 * (B - C) + D - A)));
}
__device__ void createSquareDEVICE(unsigned char square[16][3], const unsigned char *img, int x, int y, uint width, uint height, uint channels)
{

    for (int i = -1; i < 3; ++i)
    {
        for (int j = -1; j < 3; ++j)
        {
            if (x - i < 0 || y - j < 0 || x + i >= width || y + j >= height)
                continue;
            for (int k = 0; k < channels; ++k)
                square[(i + 1) + (j + 1) * 4][k] = img[channels * (x + i + (y + j) * width) + k];
        }
    }

}
__device__ double cubicInterpolateDEVICE(double A, double B, double C, double D, double t)
{
    double a = -A / 2.0f + (3.0f * B) / 2.0f - (3.0f * C) / 2.0f + D / 2.0f;
    double b = A - (5.0f * B) / 2.0f + 2.0f * C - D / 2.0f;
    double c = -A / 2.0f + C / 2.0f;
    double d = B;
    return a * t * t * t + b * t * t + c * t + d;
}


__global__ void bilinearUpscaleCUDAShared(const unsigned char *imgIn, unsigned char *imgOut, uint width, uint height, uint channels, int factor)
{
    int absX = (int) (threadIdx.x + blockIdx.x * blockDim.x);
    int absY = (int) (threadIdx.y + blockIdx.y * blockDim.y);
    uint oWidth = width * factor;
    uint oHeight = height * factor;

    if (absX >= oWidth || absY >= oHeight)
        return;

    int relX = (int) threadIdx.x;
    int relY = (int) threadIdx.y;

    uint sSize = ((uint) (8 + factor - 1) / factor) + 1;
    extern __shared__ unsigned char shared[];

    uint oldX = absX / factor;
    uint oldY = absY / factor;

    if (relX == 0 && relY == 0)
    {
        for (int i = 0; i < sSize; ++i)
            for (int j = 0; j < sSize; ++j)
            {
                if (oldX + i >= width || oldY + j >= height)
                    continue;

                for (int k = 0; k < channels; ++k)
                    shared[(i + j * sSize) * channels + k] = imgIn[(oldX + i + (oldY + j) * width) * channels + k];
            }
    }
    __syncthreads();


    int x;
    int y;
    int p00;
    int p01;
    int p10;
    int p11;
    double alpha;
    double beta;

    x = relX / factor;
    y = relY / factor;
    alpha = ((double) relX / factor) - x;
    beta = ((double) relY / factor) - y;

    for (int k = 0; k < channels; k++)
    {
        p00 = shared[(x + y * sSize) * channels + k];
        p01 = oldX + x + 1 >= width ? p00 : shared[(x + 1 + y * sSize) * channels + k];
        p10 = oldY + y + 1 >= height ? p00 : shared[(x + (y + 1) * sSize) * channels + k];
        p11 = oldX + x + 1 >= width || oldY + y + 1 >= height ? p00 : shared[(x + 1 + (y + 1) * sSize) * channels + k];

        imgOut[(absX + absY * oWidth) * channels + k] = (int) ((1 - alpha) * (1 - beta) * p00 + (1 - alpha) * beta * p01 + alpha * (1 - beta) * p10 + alpha * beta * p11);
    }

}
unsigned char *upscaleCudaBilinearShared(const unsigned char *imgIn, uint width, uint height, uint channels, int factor, uint *oWidth, uint *oHeight)
{

    //host
    uint wf = width * factor;
    uint hf = height * factor;

    uint iSize = width * height * channels;
    uint oSize = wf * hf * channels;

    auto h_imgOut = (unsigned char *) malloc(oSize * sizeof(unsigned char));
    if (h_imgOut == nullptr)
    {
        fprintf(stderr, RED "Error: " RESET "Errore nell'allocazione della memoria\n");
        return nullptr;
    }
    mlock(imgIn, iSize * sizeof(unsigned char));

    //device
    unsigned char *d_imgIn;
    unsigned char *d_imgOut;
    hipMalloc(&d_imgIn, iSize * sizeof(unsigned char));
    hipMalloc(&d_imgOut, oSize * sizeof(unsigned char));

    //copy
    hipMemcpy(d_imgIn, imgIn, iSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    //upscale
    dim3 gridSize = {(wf + 7) / 8, (hf + 7) / 8, 1};
    dim3 blockSize = {8, 8, 1};
    size_t sharedDim = (size_t) pow((int) ((8 + factor - 1) / factor) + 1, 2) * channels;
    bilinearUpscaleCUDAShared<<<gridSize, blockSize, sharedDim>>>(d_imgIn, d_imgOut, width, height, channels, factor);

    //copy back
    hipMemcpy(h_imgOut, d_imgOut, oSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    //free
    munlock(imgIn, iSize * sizeof(unsigned char));
    hipFree(d_imgIn);
    hipFree(d_imgOut);

    *oWidth = wf;
    *oHeight = hf;

    return h_imgOut;
}

__global__ void bicubicUpscaleCUDAShared(const unsigned char *imgIn, unsigned char *imgOut, uint width, uint height, int factor, uint channels)
{
    int x = (int) (threadIdx.x + blockIdx.x * blockDim.x);
    int y = (int) (threadIdx.y + blockIdx.y * blockDim.y);

    uint widthO = width * factor;
    uint heightO = height * factor;

    uint idx = x + y * widthO;

    int i;
    int j;
    double alpha;
    double beta;
    unsigned char square[16][3];
    int pixel[3];

    if (idx < widthO * heightO * channels)
    {
        i = x / factor;
        j = y / factor;

        alpha = ((double) x / factor) - i;
        beta = ((double) y / factor) - j;


        createSquareDEVICE(square, imgIn, i, j, width, height, channels);

        for (int k = 0; k < channels; k++)
        {
            double p1 = cubicInterpolateDEVICE(square[0][k], square[1][k], square[2][k], square[3][k], alpha);
            double p2 = cubicInterpolateDEVICE(square[4][k], square[5][k], square[6][k], square[7][k], alpha);
            double p3 = cubicInterpolateDEVICE(square[8][k], square[9][k], square[10][k], square[11][k], alpha);
            double p4 = cubicInterpolateDEVICE(square[12][k], square[13][k], square[14 + k][k], square[15][k], alpha);
            double p = cubicInterpolateDEVICE(p1, p2, p3, p4, beta);

            if (p > 255)
                p = 255;
            else if (p < 0)
                p = 0;

            pixel[k] = (int) p;

        }

        imgOut[idx * channels] = pixel[0];
        imgOut[(idx * channels) + 1] = pixel[1];
        imgOut[(idx * channels) + 2] = pixel[2];
    }
}
unsigned char *upscaleCudaBicubicShared(const unsigned char *imgIn, uint width, uint height, uint channels, int factor, uint *oWidth, uint *oHeight)
{

    //host
    uint wf = width * factor;
    uint hf = height * factor;
    uint iSize = width * height * channels;
    uint oSize = wf * hf * channels;
    auto h_imgOut = (unsigned char *) malloc(oSize * sizeof(unsigned char));
    if (h_imgOut == nullptr)
    {
        fprintf(stderr, RED "Error: " RESET "Errore nell'allocazione della memoria\n");
        return nullptr;
    }
    mlock(imgIn, iSize * sizeof(unsigned char));

    //device
    unsigned char *d_imgIn;
    unsigned char *d_imgOut;
    hipMalloc(&d_imgIn, iSize * sizeof(unsigned char));
    hipMalloc(&d_imgOut, oSize * sizeof(unsigned char));

    //copy
    hipMemcpy(d_imgIn, imgIn, iSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    //upscale
    dim3 gridSize = {(wf + 7) / 8, (hf + 7) / 8, 1};
    dim3 blockSize = {8, 8, 1};
    size_t sharedDim = (size_t) pow((int) ((8 + factor - 1) / factor) + 3, 2) * channels;
    bicubicUpscaleCUDA<<<gridSize, blockSize, sharedDim>>>(d_imgIn, d_imgOut, width, height, factor, channels);


    //copy back
    hipMemcpy(h_imgOut, d_imgOut, oSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    //free
    munlock(imgIn, iSize * sizeof(unsigned char));
    hipFree(d_imgIn);
    hipFree(d_imgOut);

    *oWidth = wf;
    *oHeight = hf;

    return h_imgOut;
}

